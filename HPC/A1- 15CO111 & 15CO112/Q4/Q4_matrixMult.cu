#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#define N 10
int q=1;
void printMatrix(int*,int);
__global__ void MatrixAddKernel(int*,int*,int*,int,int);


int main()
{	
	int nrow=500;
	int ncol=500;
	 
	int mat1[nrow*ncol];
	int mat2[nrow*ncol];
	int sum[nrow*ncol];
	
	srand(time(NULL));
	
	for(int i=0;i<nrow;i++)
	for(int j=0;j<ncol;j++)
	{
		mat1[i*ncol+j]=rand()%4;
		mat2[i*ncol+j]=rand()%6;
	}

	int size=nrow*ncol*sizeof(int);
	
	int* mat1_d, *mat2_d, *sum_d;
	
	hipMalloc((void**)&mat1_d,size);
	hipMalloc((void**)&mat2_d,size);
	hipMalloc((void**)&sum_d,size);
	
	hipMemcpy(mat1_d,mat1,size,hipMemcpyHostToDevice);
	hipMemcpy(mat2_d,mat2,size,hipMemcpyHostToDevice);
	
	dim3 dimBlock(32,32);
	dim3 dimGrid(ceil(ncol/32.0),ceil(nrow/32.0));
	MatrixAddKernel<<<dimGrid,dimBlock>>>(mat1_d,mat2_d,sum_d,nrow,ncol);
	
	hipMemcpy(sum,sum_d,size,hipMemcpyDeviceToHost);
	
	printMatrix(mat1,ncol);
	printMatrix(mat2,ncol);
	printMatrix(sum,ncol);
	
	hipFree(mat1_d);
	hipFree(mat2_d);
	hipFree(sum_d);	
}

__global__ void MatrixAddKernel(int* mat1_d, int* mat2_d, int* sum_d, int nrow, int ncol)
{
	int row=blockIdx.y*blockDim.y+threadIdx.y;
	int col=blockIdx.x*blockDim.x+threadIdx.x;
	
	if(row<nrow && col<ncol)
	sum_d[row*ncol+col]=mat1_d[row*ncol+col]+mat2_d[row*ncol+col];
	
}

void printMatrix(int* mat,int ncol)
{
	if(q==3)
	{
		printf("sum matrix is :\n");
		q=0;
	}
	else
	{
		printf("array %d is :\n",q);
		q++;
	}

	for(int i=0;i<N;i++)
	{
		for(int j=0;j<N;j++)
		printf("%d ",mat[i*ncol+j]);
	    
		printf("\n");
	}
	printf("\n\n");
}
