#include "hip/hip_runtime.h"
#include<stdio.h>
#include<math.h>
__global__  void add(int *c,const int *a,const int *b){

	int i=threadIdx.x;
	c[i]=a[i]+b[i];
	}
/*__global__ void input(const int * a){
	int i=threadIdx.x+blockDim.x*blockIdx.x;
	hiprand_init(1234,i,0,&a[i]);
	}*/
int main(){
int arraySize;
int a[100000],b[100000],c[100000];
printf("\nEnter Size of Arryay :");
scanf("%d",&arraySize);
for(int i=0;i<arraySize;i++){
a[i]=10*sin(i);
b[i]=10*cos(i);
}
printf("Array-1:  \n");
for(int i=0;i<arraySize;i++){
printf("%d ",a[i]);
}	
printf("\nArray-2 \n");
for(int i=0;i<arraySize;i++){
printf("%d ",b[i]);
}
	int *a_in=0;
	int *b_in=0;
	int *c_out=0;
/*	int arraySize;
	printf("\nEnter ArraySize : ");
	scanf("%d",&arraySize);
	hipMalloc((void **)&a_in,arraySize*sizeof(int));
	input<<<1,arraySize>>>(a_in);
*/	
	hipMalloc((void **)&a_in,arraySize*sizeof(int));
	hipMemcpy(a_in,a,arraySize*sizeof(int),hipMemcpyHostToDevice);
	hipMalloc((void **)&b_in,arraySize*sizeof(int));
	hipMemcpy(b_in,b,arraySize*sizeof(int),hipMemcpyHostToDevice);
	hipMalloc((void **)&c_out,arraySize*sizeof(int));
	hipMemcpy(c_out,c,arraySize*sizeof(int),hipMemcpyHostToDevice);
	add<<<1,arraySize>>>(c_out,a_in,b_in);
	hipMemcpy(c,c_out,arraySize*sizeof(int),hipMemcpyDeviceToHost);
	hipFree(a_in);
	hipFree(b_in);
	hipFree(c_out);
	hipDeviceReset();
	printf("\nResultant Array : ");
	for(int i=0;i<arraySize;i++){
		printf("%d ",c[i]);
}
printf("\n");
	return 0;
	
}
