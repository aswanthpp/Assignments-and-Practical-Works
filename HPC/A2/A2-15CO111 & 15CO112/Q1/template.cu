#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

int main(int argc, char *argv[]) {

  float *hostInput1 = nullptr;
  float *hostInput2 = nullptr;
  float *hostOutput = nullptr;
  int inputLength;

  /* parse the input arguments */
  if(argc != 4 && argc != 5)
  {
	printf("Input parameters should be of the form:\nExpectedFile.raw InputFile1.raw InputFile2.raw [Output.raw]\n");
	return 1;
  }
  
  FILE *exp = fopen(argv[1], "r");
  FILE *inp1 = fopen(argv[2], "r");
  FILE *inp2 = fopen(argv[3], "r");
  
  // Import host input data
  //@@ Read data from the raw files here
  fscanf(inp1, "%d", &inputLength);
  fscanf(inp2, "%d", &inputLength);
  //@@ Insert code here
  hostInput1 = new float[inputLength];
  hostInput2 = new float[inputLength];
  for(int i = 0; i < inputLength; ++i)
  {
	fscanf(inp1, "%f", &hostInput1[i]);
	fscanf(inp2, "%f", &hostInput2[i]);
  }

  // Declare and allocate host output
  //@@ Insert code here
  hostOutput = new float[inputLength];
  // Declare and allocate thrust device input and output vectors
  //@@ Insert code here
  thrust::device_vector<float> deviceInput1 (inputLength);
  thrust::device_vector<float> deviceInput2 (inputLength);
  thrust::device_vector<float> deviceOutput (inputLength);
  // Copy to device
  //@@ Insert code here 
  thrust::copy(hostInput1, hostInput1 + inputLength, deviceInput1.begin());
  thrust::copy(hostInput2, hostInput2 + inputLength, deviceInput2.begin());
  // Execute vector addition
  //@@ Insert Code here
  thrust::transform(deviceInput1.begin(), deviceInput1.end(), deviceInput2.begin(), deviceOutput.begin(), thrust::plus<float>());
  /////////////////////////////////////////////////////////

  // Copy data back to host
  thrust::copy(deviceOutput.begin(), deviceOutput.end(), hostOutput);
  if(argc == 5)
  {
	FILE *outp = fopen(argv[4], "w");
	fprintf(outp, "%d", inputLength);
	for(int i = 0; i < inputLength; ++i)
	{
	  fprintf(outp, "\n%.2f", hostOutput[i]);
	}
  }
  // Check if everything is valid
  bool valid = true;
  fscanf(exp, "%d", &inputLength);
  for(int i = 0; i < inputLength; ++i)
  {
	float check;
	fscanf(exp, "%f", &check);
	// Check if our answer matches up
	if(fabs(check - hostOutput[i]) > 0.001f)
	{
	  valid = false;
	  break;
	}
  }
  if(valid)
    printf("Expected output and generated output matches up\n");
  else
    printf("Expected output does not match generated output\n");
  free(hostInput1);
  free(hostInput2);
  free(hostOutput);
  return 0;
}
