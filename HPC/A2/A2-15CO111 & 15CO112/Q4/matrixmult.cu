// 15CO111 - ARVIND RAMACHANDRAN
// 15CO112 - ASWANTH P. P.

#include<iostream>
#include<assert.h>
#include"math_functions.h"
#include<hip/hip_runtime.h>


using namespace std;

__global__ void prod_arr(float* arr1,float* arr2,float* arr3,size_t M,size_t N,size_t P)
{
   size_t y=blockIdx.y*blockDim.y+threadIdx.y;
   size_t x=blockIdx.x*blockDim.x+threadIdx.x;

   if(y<M&&x<P)
   {
      float sum=0; 
      for(size_t i=0;i<N;++i)
      { 
         sum+=arr1[y*N+i]*arr2[i*P+x];
      }
      arr3[y*P+x]=sum; 
   }
}

int main()
{
   float *arr1,*arr2,*arr3;
   float *d_arr1,*d_arr2,*d_arr3;
   const size_t M=512;
   const size_t N=512;
   const size_t P=513;
   const size_t thread_count=32;
   dim3 thread_blk(thread_count,thread_count,1);
   dim3 grid((int)(M/thread_count)+1,(int)(P/thread_count)+1,1);
   
   cout<<grid.x<<","<<grid.y<<endl;
   arr1=(float *)malloc(N*M*sizeof(float));
   hipMalloc(&d_arr1,N*M*sizeof(float));
   arr2=(float *)malloc(N*P*sizeof(float));
   hipMalloc(&d_arr2,N*P*sizeof(float));
   arr3=(float *)malloc(P*M*sizeof(float));
   hipMalloc(&d_arr3,P*M*sizeof(float));

   for(size_t i=0;i<M;++i)
   {
      for(size_t j=0;j<N;++j)
      {
         arr1[i*N+j]=1;
      }
   }

   for(size_t i=0;i<N;++i)
   {
      for(size_t j=0;j<P;++j)
      {
         arr2[i*P+j]=1;
      }
   }

   hipMemcpy(d_arr1,arr1,N*M*sizeof(float),hipMemcpyHostToDevice);
   hipMemcpy(d_arr2,arr2,N*P*sizeof(float),hipMemcpyHostToDevice);

   prod_arr<<< grid,thread_blk >>>(d_arr1,d_arr2,d_arr3,M,N,P);

   hipMemcpy(arr3,d_arr3,P*M*sizeof(float),hipMemcpyDeviceToHost);
   hipMemcpy(arr1,d_arr1,N*M*sizeof(float),hipMemcpyDeviceToHost);
   hipMemcpy(arr2,d_arr2,N*P*sizeof(float),hipMemcpyDeviceToHost);

   int count=0;

   for(size_t i=0;i<M;++i)
   {
      for(size_t j=0;j<P-1;++j)
      {
         if(arr3[i*P+j]!=arr3[i*P+j+1]) 
         {  
            count++;
            cout<<i<<","<<j<<endl;
         }
      }
   }
   
   cout<<count<<endl;

   hipFree(d_arr1);
   hipFree(d_arr2);
   hipFree(d_arr3);

   free(arr1);
   free(arr2);
   free(arr3);

   cout<<"DONE\n";
   return 0;
}
