#include "hip/hip_runtime.h"
#include<stdio.h>

__global__ void addKernel( float *s, const float *a)
{
    int i = threadIdx.x;
    atomicAdd(s,a[i]);
}

int main(){
	int arraySize;
	float a[100],sum;
	printf("\nAray Size : ");
	scanf("%d",&arraySize);
	printf("\nEnter Elements :");

	for(int i=0;i<arraySize;i++){
	scanf("%f",&a[i]);
	}
	float *a_in = 0;
	float *s_out=0;
	hipMalloc((void**)&a_in, arraySize * sizeof(float));
	hipMemcpy(a_in, a, arraySize * sizeof(float), hipMemcpyHostToDevice);

	hipMalloc((void **)&s_out,sizeof(float));
	hipMemcpy(s_out,&sum,sizeof(float),hipMemcpyHostToDevice);

	addKernel<<<1,arraySize>>>(s_out,a_in);

	hipMemcpy(&sum,s_out,sizeof(float),hipMemcpyDeviceToHost);
    	hipFree(a_in);
   	hipFree(s_out);
	hipDeviceReset();
	printf("\nSum = %f\n",sum);
    return 0;
}



